
#include <hip/hip_runtime.h>
extern "C"
__global__ void runCellGeneration(int* iteration, int numberOfRows, int numberOfCols)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    int row = threadId / numberOfRows;
    int col = threadId % numberOfCols;
    int cellState = iteration[threadId];
    int aliveNeighbours = 0;

     for (int i = -1; i <= 1; i++)
     {
        for (int j = -1; j <= 1; j++)
        {
                if ((i + row < 0 || i + row > numberOfRows - 1) ||
                    (i == 0 && j == 0) ||
                    (j + col < 0 || j + col > numberOfCols - 1))
                    {
                        continue;
                    }
                    aliveNeighbours += iteration[j + col + (i + row) * numberOfRows];
        }
     }

    __syncthreads();
    
    if(cellState == 1 && (aliveNeighbours < 2 || aliveNeighbours > 3))
    {
        iteration[threadId] = 0;
    }
    else if(cellState == 0 && aliveNeighbours == 3)
    {
        iteration[threadId] = 1;
    }
}